#include <vector>

#include "caffe/layers/merge_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void MergeDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

 int num_tiles = top.size() - 1;


  bool * copy_mask = new bool[num_tiles];
  int * rand_indices = new int[num_tiles];
  int copy_size = bottom[0]->count(1);
  //printf("\n **************  %i, %i, %i *************", copy_size, bottom[0]->num(), num_tiles  );
  Dtype * sub_num_top = top[top.size()-1]->mutable_gpu_data();
  for( int n = 0; n < bottom[0]->num(); n++ )
  {
  	Dtype num_sub = rand()%max_substitute_ + min_substitute_;
  	for( int i = 0; i < num_tiles;  i++ ) { rand_indices[i] = -1;  copy_mask[i] = false; }

  	for( int i = 0; i < num_sub; i++ )
  	{
          	int ind = 0;
         	 while(1) {
                   	ind = rand()%num_tiles;
                	  if( copy_mask[ind] == false ){copy_mask[ind] = true; break;}
         	 }
          	int ind2;
          	bool is_fine = false;
         	 while(!is_fine) {
                   	is_fine = true;
                   	ind2 = rand()%num_tiles;
                	   for(int j = 0; j < num_tiles; j++ ) if(rand_indices[j]==ind2){ is_fine=false; break; }
          	}
        	  rand_indices[ind] = ind2;
  	}
	CUDA_CHECK(hipMemcpy(sub_num_top+n, &num_sub, sizeof(Dtype) ,hipMemcpyHostToDevice)); //sub_num_top[n] = num_sub;
 	 for (int i = 0; i < num_tiles; ++i) {
	 if( copy_mask[i] == false )
                	CUDA_CHECK(hipMemcpy( (Dtype*)top[i]->gpu_data()+n*copy_size, (Dtype*)bottom[i]->gpu_data() + n*copy_size,
				sizeof(Dtype)*copy_size, hipMemcpyDeviceToDevice ));
        	else
                        CUDA_CHECK(hipMemcpy( (Dtype*)top[i]->gpu_data()+n*copy_size, (Dtype*)bottom[num_tiles + rand_indices[i]]->gpu_data() + n*copy_size, 
                                sizeof(Dtype)*copy_size, hipMemcpyDeviceToDevice ));
  	}
        
  }
  delete copy_mask;
  delete rand_indices;
}

INSTANTIATE_LAYER_GPU_FUNCS(MergeDataLayer);

}  // namespace caffe
